/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

void printTensorShape(const torch::Tensor& tensor, const std::string& name) {
    std::cout << "Shape of " << name << ": ";
    for (const auto& dim : tensor.sizes()) {
        std::cout << dim << " ";
    }
    std::cout << std::endl;
}


std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,             // 3
	const torch::Tensor& means3D,                // 182686, 3
    const torch::Tensor& colors,                 // 0     colors_precomp
    const torch::Tensor& opacity,				 // 182686, 1
	const torch::Tensor& scales,                 // 182686, 3
	const torch::Tensor& rotations,              // 182686, 4
	const float scale_modifier,                  // 1
	const torch::Tensor& cov3D_precomp,          // 0
	const torch::Tensor& viewmatrix,             // 4, 4
	const torch::Tensor& projmatrix,             // 4, 4
	const float tan_fovx,                        // 0.844697
	const float tan_fovy,                        // 0.467948
    const int image_height,						 // 545
    const int image_width,                       // 980
	const torch::Tensor& sh,                     // 182686, 16, 3
	const int degree,                            // 0
	const torch::Tensor& campos,                 // 3
	const bool prefiltered,                      // 0
	const bool debug)                            // 0
{
	// printTensorShape(background, "background");
    // printTensorShape(means3D, "means3D");
    // printTensorShape(colors, "colors");
	// printTensorShape(opacity, "opacity");
	// printTensorShape(scales, "scales");
    // printTensorShape(rotations, "rotations");
    // std::cout << "scale_modifier: " << scale_modifier << std::endl;
	// printTensorShape(cov3D_precomp, "cov3D_precomp");
	// printTensorShape(viewmatrix, "viewmatrix");
    // printTensorShape(projmatrix, "projmatrix");
    // std::cout << "tan_fovx: " << tan_fovx << std::endl;
    // std::cout << "tan_fovy: " << tan_fovy << std::endl;
    // std::cout << "image_height: " << image_height << std::endl;
    // std::cout << "image_width: " << image_width << std::endl;
	// printTensorShape(sh, "sh");
    // std::cout << "degree: " << degree << std::endl;
	// printTensorShape(campos, "campos");
    // std::cout << "prefiltered: " << prefiltered << std::endl;
    // std::cout << "debug: " << debug << std::endl;
	// std::cin.get();
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);    // 182686
  const int H = image_height;       // 545
  const int W = image_width;		// 980
  
  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);  // NUM_CHANNELS = 3 for RGB
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte); // uint8_t
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  // 这些函数对象的主要目的是调整对应的Tensor的大小，并返回一个指向Tensor数据的指针。
  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);    // 16
      }

	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,           // 182686 3 
	const torch::Tensor& radii,
    const torch::Tensor& colors,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color,      // shape : [3, 545, 980]
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,                            // 1570996   (num_rendered)
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool debug) 
{
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);       // 545
  const int W = dL_dout_color.size(2);       // 980
  
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, M, R,
	  background.contiguous().data<float>(),
	  W, H, 
	  means3D.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
	  colors.contiguous().data<float>(),
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<float>(),
	  dL_dmeans2D.contiguous().data<float>(),
	  dL_dconic.contiguous().data<float>(),  
	  dL_dopacity.contiguous().data<float>(),
	  dL_dcolors.contiguous().data<float>(),
	  dL_dmeans3D.contiguous().data<float>(),
	  dL_dcov3D.contiguous().data<float>(),
	  dL_dsh.contiguous().data<float>(),
	  dL_dscales.contiguous().data<float>(),
	  dL_drotations.contiguous().data<float>(),
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}